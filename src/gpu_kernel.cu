#include "hip/hip_runtime.h"

#include "common.h"


__global__ void GaussianFilter(unsigned char * img_in , unsigned char * img_out , int width , int height , int channels , int * gaussian_filter)
{
	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;


	if (col < width && row < height)
	{
		int gaussianSum , min_row , max_row , min_col , max_col , g_x , g_y , imageIndex;
		for (int channel = 0; channel < channels; channel++)
	    {
	        imageIndex = (row * width + col) * channels + channel;
	        gaussianSum = 0;
	        min_row = row - GAUSSIAN_DIM / 2 > 0 ? row - GAUSSIAN_DIM / 2 : 0;
	        max_row = (row + GAUSSIAN_DIM / 2 + 1) < height ? (row + GAUSSIAN_DIM / 2 + 1) :  height;
	        min_col = col - GAUSSIAN_DIM / 2 > 0 ? col - GAUSSIAN_DIM / 2 : 0;
	        max_col = (col + GAUSSIAN_DIM / 2 + 1) < width ? (col + GAUSSIAN_DIM / 2 + 1) : width;

	        g_x = 0;
	        for (int offX = min_row; offX < max_row; offX++)
	        {
	            g_y = 0;
	            for (int offY = min_col; offY < max_col; offY++)
	            {
	                gaussianSum += img_in[(offX * width + offY) * channels + channel] * gaussian_filter[g_x*GAUSSIAN_DIM + g_y];
	                g_y++;
	            }
	            g_x++;
	        }
		
	        img_out[imageIndex] = gaussianSum / 273;
	    }
	}
}

__global__ void RGB2GRAY(unsigned char * img_in , unsigned char * img_out , int width , int height , int channels)
{
	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;

	if (col < width && row < height)
	{
		int index = (row*width + col)*channels;
		img_out[row*width + col] = img_in[index] / 3 + img_in[index + 1] / 3 + img_in[index + 2] / 3;
	}
}

__global__ void InitContour(double * contour , int width , int height)
{
	int col = threadIdx.x + blockDim.x*blockIdx.x;
	int row = threadIdx.y + blockDim.y*blockIdx.y;

	if (col < width && row < height)
	{
		int x = double(row) - height / 2.0;
        int y = double(col) - width / 2.0;
        contour[row * width + col] = 900.0 / (900.0 + x * x + y * y) - 0.5; //radius/(radius + x*x + y*y) - 0.5;
    }
}


void Preprocess_kernel(Image &img_in, Image &img_out){
	img_out.channels = img_in.channels;
	img_out.height = img_in.height;
	img_out.width = img_in.width;
	img_out.size = img_in.size;
	img_out.contour0 = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.contour = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.contourOld = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.img = (unsigned char *)malloc(sizeof(unsigned char) * img_out.size);
	memcpy(img_out.img, img_in.img, sizeof(unsigned char) * img_in.size);

    int size = img_in.height * img_in.width * img_in.channels;
    unsigned char *d_img_in , *d_img_out , *d_img_flatten; double *d_img_contour;
    int *gaussian_filter, *d_gaussian_filter;
    
    gaussian_filter = (int * )malloc(sizeof(int) * GAUSSIAN_DIM * GAUSSIAN_DIM);
    for (int i=0;i<GAUSSIAN_DIM; i++){
		for(int j=0;j<GAUSSIAN_DIM; j++){
			gaussian_filter[i*GAUSSIAN_DIM + j] = GAUSSIAN[i][j];
		}
    }

    hipMalloc((void**)&d_img_in, size*sizeof(unsigned char));
    hipMalloc((void**)&d_img_out, size*sizeof(unsigned char));
    hipMalloc((void**)&d_img_flatten, img_in.height * img_in.width * sizeof(unsigned char ));
    hipMalloc((void**)&d_img_contour, img_in.height * img_in.width * sizeof(double));
    hipMalloc((void**)&d_gaussian_filter, GAUSSIAN_DIM * GAUSSIAN_DIM * sizeof(int));
    
    hipMemcpy(d_img_in, img_in.img, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_gaussian_filter, gaussian_filter, GAUSSIAN_DIM * GAUSSIAN_DIM * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_img_out, img_out.img, size * sizeof(unsigned char), hipMemcpyHostToDevice);

	
    /*************** Kernel calls**************/
	dim3 grid , block;
	block.x = BLOCK_SIZE_X;
	block.y = BLOCK_SIZE_Y;
	grid.x = (img_in.width  % block.x==0) ? img_in.width  / block.x : img_in.width  / block.x+1;
	grid.y = (img_in.height % block.y==0) ? img_in.height / block.y : img_in.height / block.y+1;

	//kernel 1
	GaussianFilter <<< grid , block >>> (d_img_in, d_img_out , img_in.width , img_in.height , img_in.channels , d_gaussian_filter);

    // kernel 2  
    if(img_out.channels>1)
	  RGB2GRAY <<< grid , block >>>(d_img_out , d_img_flatten , img_in.width , img_in.height , img_in.channels);

    //kernel 3
    InitContour <<< grid , block >>>(d_img_contour , img_in.width , img_in.height);

    hipDeviceSynchronize();

    img_out.channels = 1;
    img_out.size = img_out.height * img_out.width;
    
    hipMemcpy(img_out.img, d_img_flatten, img_out.size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    img_out.img = (unsigned char *)realloc(img_out.img, sizeof(unsigned char) * img_out.size);
    hipMemcpy(img_out.contour0, d_img_contour, img_out.size * sizeof(double), hipMemcpyDeviceToHost);
	
    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_img_flatten);
    hipFree(d_img_contour);
    hipFree(d_gaussian_filter);
    free(gaussian_filter);
}

