#include "hip/hip_runtime.h"

#include "common.h"

__global__ void GaussianFilter(unsigned char *img_in, unsigned char *img_out, int width, int height, int channels, int *gaussian_filter)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < width && row < height)
	{
		int gaussianSum, min_row, max_row, min_col, max_col, g_x, g_y, imageIndex;
		for (int channel = 0; channel < channels; channel++)
		{
			imageIndex = (row * width + col) * channels + channel;
			gaussianSum = 0;
			min_row = row - GAUSSIAN_DIM / 2 > 0 ? row - GAUSSIAN_DIM / 2 : 0;
			max_row = (row + GAUSSIAN_DIM / 2 + 1) < height ? (row + GAUSSIAN_DIM / 2 + 1) : height;
			min_col = col - GAUSSIAN_DIM / 2 > 0 ? col - GAUSSIAN_DIM / 2 : 0;
			max_col = (col + GAUSSIAN_DIM / 2 + 1) < width ? (col + GAUSSIAN_DIM / 2 + 1) : width;

			g_x = 0;
			for (int offX = min_row; offX < max_row; offX++)
			{
				g_y = 0;
				for (int offY = min_col; offY < max_col; offY++)
				{
					gaussianSum += img_in[(offX * width + offY) * channels + channel] * gaussian_filter[g_x * GAUSSIAN_DIM + g_y];
					g_y++;
				}
				g_x++;
			}

			img_out[imageIndex] = gaussianSum / 273;
		}
	}
}

__global__ void RGB2GRAY(unsigned char *img_in, unsigned char *img_out, int width, int height, int channels)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < width && row < height)
	{
		int index = (row * width + col) * channels;
		img_out[row * width + col] = img_in[index] / 3 + img_in[index + 1] / 3 + img_in[index + 2] / 3;
	}
}

__global__ void InitContour(double *contour, int width, int height)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < width && row < height)
	{
		int x = double(row) - height / 2.0;
		int y = double(col) - width / 2.0;
		contour[row * width + col] = 900.0 / (900.0 + x * x + y * y) - 0.5; //radius/(radius + x*x + y*y) - 0.5;
	}
}

void Preprocess_kernel(Image &img_in, Image &img_out)
{
	img_out.channels = img_in.channels;
	img_out.height = img_in.height;
	img_out.width = img_in.width;
	img_out.size = img_in.size;
	img_out.contour0 = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.contour = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.contourOld = (double *)malloc(sizeof(double) * img_out.width * img_out.height);
	img_out.img = (unsigned char *)malloc(sizeof(unsigned char) * img_out.size);
	memcpy(img_out.img, img_in.img, sizeof(unsigned char) * img_in.size);

	int size = img_in.height * img_in.width * img_in.channels;
	unsigned char *d_img_in, *d_img_out, *d_img_flatten;
	double *d_img_contour;
	int *gaussian_filter, *d_gaussian_filter;

	gaussian_filter = (int *)malloc(sizeof(int) * GAUSSIAN_DIM * GAUSSIAN_DIM);
	for (int i = 0; i < GAUSSIAN_DIM; i++)
	{
		for (int j = 0; j < GAUSSIAN_DIM; j++)
		{
			gaussian_filter[i * GAUSSIAN_DIM + j] = GAUSSIAN[i][j];
		}
	}

	hipMalloc((void **)&d_img_in, size * sizeof(unsigned char));
	hipMalloc((void **)&d_img_out, size * sizeof(unsigned char));
	hipMalloc((void **)&d_img_flatten, img_in.height * img_in.width * sizeof(unsigned char));
	hipMalloc((void **)&d_img_contour, img_in.height * img_in.width * sizeof(double));
	hipMalloc((void **)&d_gaussian_filter, GAUSSIAN_DIM * GAUSSIAN_DIM * sizeof(int));

	hipMemcpy(d_img_in, img_in.img, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_gaussian_filter, gaussian_filter, GAUSSIAN_DIM * GAUSSIAN_DIM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_img_out, img_out.img, size * sizeof(unsigned char), hipMemcpyHostToDevice);

	/*************** Kernel calls**************/
	dim3 grid, block;
	block.x = BLOCK_SIZE_X;
	block.y = BLOCK_SIZE_Y;
	grid.x = (img_in.width % block.x == 0) ? img_in.width / block.x : img_in.width / block.x + 1;
	grid.y = (img_in.height % block.y == 0) ? img_in.height / block.y : img_in.height / block.y + 1;

	//kernel 1
	GaussianFilter<<<grid, block>>>(d_img_in, d_img_out, img_in.width, img_in.height, img_in.channels, d_gaussian_filter);

	// kernel 2
	if (img_out.channels > 1)
		RGB2GRAY<<<grid, block>>>(d_img_out, d_img_flatten, img_in.width, img_in.height, img_in.channels);

	//kernel 3
	InitContour<<<grid, block>>>(d_img_contour, img_in.width, img_in.height);

	hipDeviceSynchronize();

	img_out.channels = 1;
	img_out.size = img_out.height * img_out.width;

	hipMemcpy(img_out.img, d_img_flatten, img_out.size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	img_out.img = (unsigned char *)realloc(img_out.img, sizeof(unsigned char) * img_out.size);
	hipMemcpy(img_out.contour0, d_img_contour, img_out.size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_img_in);
	hipFree(d_img_out);
	hipFree(d_img_flatten);
	hipFree(d_img_contour);
	hipFree(d_gaussian_filter);
	free(gaussian_filter);
}

__global__ void GetAverageIntensityOfRegionsKernel(Image img, double *n1, double *n2, double *d1, double *d2)
{

	// Calculate threadId and perform computation
	int tidX = threadIdx.x + blockIdx.x * blockDim.x;
	int tidY = threadIdx.y + blockIdx.y * blockDim.y;
	int gIndex = tidY * img.width + tidX;

	if (gIndex >= img.size)
		return;

	double H_phi = 0.5 * (1 + (2 / PI) * atan(img.contour[gIndex] / H));


	atomicAdd(n1, ((double)img.img[gIndex] * H_phi));
	atomicAdd(d1, H_phi);
	atomicAdd(n2, ((double)img.img[gIndex]) * (1 - H_phi));
	atomicAdd(d2, 1 - H_phi);
}

__device__ double pow(double x, int p)
{
	return x * x;
}

__global__ void ChanVeseCoreKernel(Image img, double *avgIntensity)
{
	// Calculate identify threads
	int i = threadIdx.y + blockIdx.y * blockDim.y; // i
	int j = threadIdx.x + blockIdx.x * blockDim.x; // j
	// int gIndex = i * img.width + j;

	// Image boundary safety
	if (j < 1 || j >= img.width - 1 || i < 1 || i >= img.height - 1)
		return;
	// Calculate Avg Intensity of Regions
	double c1 = avgIntensity[0] / avgIntensity[2];
	double c2 = avgIntensity[1] / avgIntensity[3];

	// if (i == 1 && j == 1)
	// 	printf("avg-Intensity: %f %f\n", c1, c2);
	double i_j = img.contour0[i * img.width + j];
	double iPlus_j = img.contour0[(i + 1) * img.width + j];
	double i_jPlus = img.contour0[i * img.width + j + 1];
	double i_jMinus = img.contour0[i * img.width + j - 1];
	double iMinus_j = img.contour0[(i - 1) * img.width + j];
	double iMinus_jPlus = img.contour0[(i - 1) * img.width + j + 1];
	double iMinus_jMinus = img.contour0[(i - 1) * img.width + j - 1];
	double iPlus_jMinus = img.contour0[(i + 1) * img.width + j - 1];

	double L = 1;
	double C1 = 1 / sqrt(EPSILON +
						 pow((iPlus_j - i_j), 2) +
						 pow((i_jPlus - i_jMinus), 2) / 4);

	double C2 = 1 / sqrt(EPSILON +
						 pow((i_j - iMinus_j), 2) +
						 pow((iMinus_jPlus - iMinus_jMinus), 2) / 4);

	double C3 = 1 / sqrt(EPSILON +
						 pow((iPlus_j - iMinus_j), 2) / 4.0 +
						 pow((i_jPlus - i_j), 2));

	double C4 = 1 / sqrt(EPSILON +
						 pow((iPlus_jMinus - iMinus_jMinus), 2) / 4.0 +
						 pow((iPlus_j - iPlus_jMinus), 2));

	double delPhi = H / (PI * (H * H + (i_j) * (i_j)));
	double Multiple = DT * delPhi * MU * (double(P) * pow(L, P - 1));
	double F = H / (H + Multiple * (C1 + C2 + C3 + C4));
	Multiple = Multiple / (H + Multiple * (C1 + C2 + C3 + C4));
	double F1 = Multiple * C1;
	double F2 = Multiple * C2;
	double F3 = Multiple * C3;
	double F4 = Multiple * C4;

	double CurrPixel = i_j - DT * delPhi * (NU + lambda1 * pow(img.img[i * img.width + j] - c1, 2) - lambda2 * pow(img.img[i * img.width + j] - c2, 2));
	img.contour[i * img.width + j] = F1 * iPlus_j +
									 F2 * iMinus_j +
									 F3 * i_jPlus +
									 F4 * i_jMinus + F * CurrPixel;
}

void GetAverageIntensityOfRegions(dim3 grid, dim3 block, Image d_img, double *avgIntensity)
{
	GetAverageIntensityOfRegionsKernel<<<grid, block>>>(d_img, avgIntensity, avgIntensity + 1, avgIntensity + 2, avgIntensity + 3);
}

void ChanVeseCore(dim3 grid, dim3 block, Image &img, double *avgIntensity)
{
	ChanVeseCoreKernel<<<grid, block>>>(img, avgIntensity);
}
